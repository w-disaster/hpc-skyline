#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-skyline.cu - Skyline set computation with CUDA
 *
 * Author: Fabri Luca 
 * Serial Number: 0000892878
 * Email: luca.fabri@studio.unibo.it
 * 
 * ---------------------------------------------------------------------------
 * 
 * Skyline set computation with CUDA. 
 * Given P a set of N points with dimension D, p1, p2 two points in P,
 * we say that p1 dominates p2 if:
 * - for each dimension k: p1[k] >= p2[k] , 0 <= k < D;
 * - exists at least one dimension j such that: p1[j] > p2[j] , 0 <= j < D.
 *
 * The Skyline set is, for definition, composed of all points in P that aren't
 * dominated by other points in P.
 *
 * Compile with:
 * nvcc -o cuda-skyline --gpu-architecture 30 --compute_30 cuda-skyline.cu
 * Or from Makefile:
 * make cuda
 *
 * Run with:
 * ./cuda-skyline < input_file > output_file
 *
 * Please not that the input_file provided as argument must contains:
 * - The dimension of the points, in the first line. Next chars, if present, 
 *   are ignored;
 * - The number of points in the 2nd line;
 * - The input set P whose points are in separed rows and each dimension value 
 *   separated by space or tab.
 *
 ****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include "lib/hpc.h"

#define LINE_LENGHT 4000
#define WARP_SIZE 32

/* This function reads the points from a file descriptor and saves
 * them in the return matrix. Also, it stores the dimension D and
 * the number of points N onto two int memory locations.
 */
double* read_points(FILE* fd, int* N, int* D){
    char line[LINE_LENGHT];
    const size_t BUF_SIZE = sizeof(line);
	    
    /* Read the dimension: fetch the first line until space */
    char* dim;
    dim = fgets(line, BUF_SIZE, fd);
    sscanf(dim, "%d", D);
    
    /* Read the number of points: fetch the second line until newline */
    char* n;
    n = fgets(line, BUF_SIZE, fd);
    sscanf(n, "%d", N);
    
    /* Allocate the matrix (N x D), where each line i contains the values
	   of the points on that dimension i.
	*/
    double *matrix = (double*) malloc((*N) * (*D) * sizeof(double));
	
    char* str;
    const char* s = " ";
    char* token;
    char* ptr;
    for(int i = 0; i < *N; i++){
        /* Read current line */
        str = fgets(line, BUF_SIZE, fd);
        /* Split the string read on s=" " separator and fetch the values */
        token = strtok(str, s);
        for(int k = 0; k < *D && token != NULL; k++){
            /* convert ASCII string to doubleing-point number */
            matrix[k * (*N) + i] = strtod(token, &ptr);
            token = strtok(NULL, s);
        }
    }
    return matrix;
}

/* Returns true if the array s dominates the array d. 
 * Parameters:
 * - s, d: arrays of double
 * - length: number of elements of s and d
 * - offset: distance between two elements that we must read in array s, d
 */
__device__ bool dominance(double *s, double *d, int length, int offset){
    bool strictly_minor = false;
    bool strictly_major = false;
    /* Iterate over each index: 
     * if s[i] < d[i] then s doesn't dominate d --> exit from loop and return */
    for(int i = 0; i < length && !strictly_minor; i++){
        if(s[i * offset] < d[i * offset]){
			 strictly_minor = true;
		}
        if(s[i * offset] > d[i * offset]){
			strictly_major = true;
		}
    }
    /* If there aren't elements strictly minor and exist at least on element
     * strictly major then s dominates d
     */
    return !strictly_minor && strictly_major;
}

/* Kernel function:
 * each thread has the purpose to determine if the number in charge is in
 * the Skyline set. To do so, this function iterates on all the points and stops
 * if any of them dominates it.
 * The result, in the end, is put in the array S, stored in the global memory. 
 */
__global__ void compute_skyline(double *points, bool *S, int *k, int n, int d){
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y < n){
		bool is_skyline_point = true;
		for(int i = 0; i < n && is_skyline_point; i++){
			/* If num is dominates by another number then it is not
			   in the Skyline set
			*/
			if(i != y){
				if(dominance(&points[i], &points[y], d, n)){
					is_skyline_point = false;						 
				}
			}
		}
		/* Copy the results on the device global memory */
		S[y] = is_skyline_point;
        atomicAdd(k, is_skyline_point); 
	}
}

/*
 * This function prints to the file descriptor fd given as parameter:
 * - The dimension D of the points;
 * - The cardinality K of the Skyline set;
 * - The Skyline set.
 */
void print_skyline(FILE* fd, bool *S, double *points, int N, int D, int K){
    int i, j;
    /* Print D, K */
    fprintf(fd, "%d\n%d\n", D, K);

    /* Print the Skyline set */
    for(i = 0; i < N; i++){
        if(S[i]){
            for(j = 0; j < D; j++){
                fprintf(fd, "%lf ", points[j * N + i]);
            }
            fprintf(fd, "\n");
        }
    }
}

int main(int argc, char* argv[]){
	double t_start = hpc_gettime();
   	/* Allocate memory to store the number of points, them dimension and the points */
	int* D = (int*) malloc(sizeof(int));
    int* N = (int*) malloc(sizeof(int));
    double* points = read_points(stdin, N, D);

	/* - Define the matrix dimension, 
	   - Allocate space on the device global memory 
	   - Copy the array points on the allocated space
	 */
	const size_t size = (*N) * (*D) * sizeof(double);
    double* d_points;
	cudaSafeCall(hipMalloc((void**)&d_points, size));
	cudaSafeCall(hipMemcpy(d_points, points, size, hipMemcpyHostToDevice));

	/* Allocate space where the kernel function will store the result */
	bool *S, *d_S;
	cudaSafeCall(hipMalloc((void**)&d_S, (*N) * sizeof(bool)));

	/* Allocate space in order to store the cardinality of the Skyline set */
    int *K, *d_K;
    K = (int*) malloc(sizeof(int));
    *K = 0;
    cudaSafeCall(hipMalloc((void**)&d_K, sizeof(int)));    
    cudaSafeCall(hipMemcpy(d_K, K, sizeof(int), hipMemcpyHostToDevice));

	/* Define the block and grid dimensions */
	dim3 block(1, WARP_SIZE * 2);
	dim3 grid(1, ((*N) + WARP_SIZE * 2 - 1)/(WARP_SIZE * 2));
		
	/* - Kernel function call to determine the Skyline set
	   - Calculate the time spent 
	   - Check for errors occurred in the GPU
	*/

	compute_skyline<<<grid, block>>>(d_points, d_S, d_K, *N, *D);
	
	/* While Kernel function is executing on device, allocate memory on heap 
	 * in order to store the result 
     */
	S = (bool*) malloc((*N) * sizeof(bool));

	/* Wait the Kernel to finish and check errors */
	cudaCheckError();	
		
	/* - Copy the result from device memory to host's
       - Copy the Skyline cardinality from device to host memory
	   - Print the points in the Skyline set 
	*/
	cudaSafeCall(hipMemcpy(S, d_S, (*N) * sizeof(bool), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(K, d_K, sizeof(int), hipMemcpyDeviceToHost));
    print_skyline(stdout, S, points, *N, *D, *K);

	/* Free space on device and host heap memory */
	hipFree(d_points);
    hipFree(d_K);
	free(points);
	free(S);
	free(D);
	free(N);
    free(K);

	/* Print the time spent  by the kernel to determine the Skyline set */
	fprintf(stdout, "%lf\n", hpc_gettime() - t_start);	
    return 0;
}
