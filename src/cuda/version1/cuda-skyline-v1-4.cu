#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-skyline.cu - Skyline set computation with CUDA
 *
 * Author: Fabri Luca 
 * Serial Number: 0000892878
 * Email: luca.fabri@studio.unibo.it
 * 
 * ---------------------------------------------------------------------------
 * 
 * Skyline set computation with CUDA. 
 * Given P a set of N points with dimension D, p1, p2 two points in P,
 * we say that p1 dominates p2 if:
 * - for each dimension k: p1[k] >= p2[k] , 0 <= k < D;
 * - exists at least one dimension j such that: p1[j] > p2[j] , 0 <= j < D.
 *
 * The Skyline set is, for definition, composed of all points in P that aren't
 * dominated by other points in P.
 *
 * Compile with:
 * nvcc -o cuda-skyline --gpu-architecture compute_30 cuda-skyline.cu
 * Or from Makefile:
 * make cuda
 *
 * Run with:
 * ./cuda-skyline < input_file > output_file
 *
 * Please not that the input_file provided as argument must contains:
 * - The dimension of the points, in the first line. Next chars, if present, 
 *   are ignored;
 * - The number of points in the 2nd line;
 * - The input set P whose points are in separed rows and each dimension value 
 *   separated by space or tab.
 *
 ****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include "lib/hpc.h"

#define LINE_LENGHT 4000
#define WARP_SIZE 32

/* 
 * This function reads the points from a file descriptor and saves
 * them into an array. Also, it stores the dimension D and
 * the number of points N onto two int memory locations.
 * 
 * Parameters:
 * - fd: file descriptor
 * - N: pointer to integer where this function stores the number of points read
 * - D: pointer to int where this function stores the dimension of the points.
 * 
 * It returns the pointer to the allocated array containing the points.
 */

double* read_points(FILE* fd, int* N, int* D){
    char line[LINE_LENGHT];
    const size_t BUF_SIZE = sizeof(line);
	    
    /* Read the dimension: fetch the first line until space */
    char* dim;
    dim = fgets(line, BUF_SIZE, fd);
    sscanf(dim, "%d", D);
    
    /* Read the number of points: fetch the second line until newline */
    char* n;
    n = fgets(line, BUF_SIZE, fd);
    sscanf(n, "%d", N);
    
    /* Allocate the matrix (N x D), where each line i contains the values
	   of the points on that dimension i.
	*/
    double *matrix = (double*) malloc((*N) * (*D) * sizeof(double));
	
    char* str;
    const char* s = " ";
    char* token;
    char* ptr;
    for(int i = 0; i < *N; i++){
        /* Read current line */
        str = fgets(line, BUF_SIZE, fd);
        /* Split the string read on s=" " separator and fetch the values */
        token = strtok(str, s);
        for(int k = 0; k < *D && token != NULL; k++){
            /* convert ASCII string to doubleing-point number */
            matrix[k * (*N) + i] = strtod(token, &ptr);
            token = strtok(NULL, s);
        }
    }
    return matrix;
}

/* Returns true if the array s dominates the array d. 
 * Parameters:
 * - s, d: arrays of double
 * - dim: number of elements of s and d
 * - offset: distance between two elements that we must read in array s, d
 */
__device__ bool dominance(double *s, double *d, int dim, int offset){
    bool strictly_major = false;
    /* Iterate over each index: 
     * if s[i] < d[i] then s doesn't dominate d --> return */
    for(int i = 0; i < dim; i++){
        if(s[i * offset] < d[i * offset]){
			 return false;
		}
        if(s[i * offset] > d[i * offset]){
			strictly_major = true;
		}
    }
    /* If there aren't elements strictly minor and exist at least on element
     * strictly major then s dominates d
     */
    return strictly_major;
}

/* Kernel function:
 * each thread has the purpose to determine if the number in charge is in
 * the Skyline set. To do so, this function iterates on all the points and stops
 * if any of them dominates it.
 * The result, in the end, is put in the array S, stored in the global memory. 
 */
__global__ void compute_skyline(double *points, bool *S, int *k, int n, int d){
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y < n){
		bool is_skyline_point = true;
		for(int i = 0; i < n && is_skyline_point; i++){
			/* If num is dominates by another number then it is not
			   in the Skyline set
			*/
			if(i != y){
				if(dominance(&points[i], &points[y], d, n)){
					is_skyline_point = false;						 
				}
			}
		}
		/* Copy the results on the device global memory */
		S[y] = is_skyline_point;
        /* Increment by 1 the cardinality of Skyline set if the number 
           is in the Skyline set, by 0 otherwise 
         */
        atomicAdd(k, is_skyline_point); 
	}
}

/*
 * This function prints to the file descriptor fd given as parameter:
 * - The dimension D of the points;
 * - The cardinality K of the Skyline set;
 * - The Skyline set.
 */
__host__ void print_skyline(FILE* fd, bool *S, double *points, int N, int D, int K){
    int i, j;
    /* Print D, K */
    fprintf(fd, "%d\n%d\n", D, K);

    /* Print the Skyline set */
    for(i = 0; i < N; i++){
        if(S[i]){
            for(j = 0; j < D; j++){
                fprintf(fd, "%f ", points[j * N + i]);
            }
            fprintf(fd, "\n");
        }
    }
}

int main(void){
/* Allocate memory to store the number of points, them dimension and the points */
	int* D = (int*) malloc(sizeof(int));
    int* N = (int*) malloc(sizeof(int));

	double* points = read_points(stdin, N, D);
   
	/* - Define the matrix dimension, 
	   - Allocate space on the device global memory 
	   - Copy the array points on the allocated space
	 */
	const size_t size = (*N) * (*D) * sizeof(double);
    double* d_points;
	cudaSafeCall(hipMalloc((void**)&d_points, size));
	cudaSafeCall(hipMemcpy(d_points, points, size, hipMemcpyHostToDevice));

	/* Allocate space where the kernel function will store the result */
	bool *S, *d_S;
	cudaSafeCall(hipMalloc((void**)&d_S, (*N) * sizeof(bool)));

	/* Allocate space in order to store the cardinality of the Skyline set */
    int *K, *d_K;
    K = (int*) malloc(sizeof(int));
    *K = 0;
    cudaSafeCall(hipMalloc((void**)&d_K, sizeof(int)));    
    cudaSafeCall(hipMemcpy(d_K, K, sizeof(int), hipMemcpyHostToDevice));

	/* Define the block and grid dimensions */
	dim3 block(1, WARP_SIZE * 2);
	dim3 grid(1, ((*N) + WARP_SIZE * 2 - 1)/(WARP_SIZE * 2));
	
	hipEvent_t t_kernel_start, t_kernel_stop;
	hipEventCreate(&t_kernel_start);
	hipEventCreate(&t_kernel_stop);	

	hipEventRecord(t_kernel_start);
	
	/* Kernel function call to determine the Skyline set */
	compute_skyline<<<grid, block>>>(d_points, d_S, d_K, *N, *D);
	
	hipEventRecord(t_kernel_stop);

	/* Wait the Kernel to finish and check errors */
	cudaCheckError();	

    /* While Kernel function is executing on device, allocate memory on heap 
	 * in order to store the result 
     */
	S = (bool*) malloc((*N) * sizeof(bool));
	
	/* - Copy the result from device memory to host's
       - Copy the Skyline cardinality from device to host memory
	   - Print the points in the Skyline set 
	*/
	cudaSafeCall(hipMemcpy(S, d_S, (*N) * sizeof(bool), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(K, d_K, sizeof(int), hipMemcpyDeviceToHost));
    print_skyline(stdout, S, points, *N, *D, *K);

	/* Free space on device and host heap memory */
	hipFree(d_points);
    hipFree(d_K);
	free(points);
	free(S);
	free(D);
	free(N);
    free(K);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, t_kernel_start, t_kernel_stop);	
	fprintf(stdout, "%f\n", milliseconds / 1000);   
	return 0;
}

