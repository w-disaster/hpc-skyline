#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include "lib/hpc.h"

#define LINE_LENGHT 4000
#define WARP_SIZE 32
#define MAX_DIM 200

/* This function reads the points from a file descriptor and saves
 * them in the return matrix. Also, it stores the dimension D and
 * the number of points N onto two int memory locations.
 */
double* build_matrix(FILE* fd, int* N, int* D){
    char line[LINE_LENGHT];
    const size_t BUF_SIZE = sizeof(line);
	    
    /* Read the dimension: fetch the first line until space */
    char* dim;
    dim = fgets(line, BUF_SIZE, fd);
    sscanf(dim, "%d", D);
    printf("%d\n", *D);
    
    /* Read the number of points: fetch the second line until newline */
    char* n;
    n = fgets(line, BUF_SIZE, fd);
    sscanf(n, "%d", N);
    printf("%d\n", *N);

    /* Allocate the matrix (D x N), where each line i contains the values
	   of the points on that dimension i.
	*/
    double *matrix = (double*) malloc((*D) * (*N) * sizeof(double));
	
    char* str;
    const char* s = " ";
    char* token;
    char* ptr;
    for(int i = 0; i < *N; i++){
        /* Read current line */
        str = fgets(line, BUF_SIZE, fd);
        /* Split the string read on s=" " separator and fetch the values */
        token = strtok(str, s);
        for(int k = 0; k < *D && token != NULL; k++){
            /* convert ASCII string to doubleing-point number */
            matrix[i * (*D) + k] = strtod(token, &ptr);
            token = strtok(NULL, s);
        }
    }
    return matrix;
}

/* Returns true if s dominates d */
__device__ bool dominance(double *s, double *d, int dim){
    bool weakly_major = true;
    bool stricly_major = false;
    for(int i = 0; i < dim && weakly_major; i++){
        if(s[i] < d[i]){
			 weakly_major = false;
		}
        if(s[i] > d[i]){
			stricly_major = true;
		}
    }
    return weakly_major && stricly_major;
}

/* Kernel function */
__global__ void skyline(double *points, bool *S, int n, int d){
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y < n){
		bool is_skyline_point = true;
		for(int i = 0; i < n && is_skyline_point; i++){
			/* If num is dominates by another number then it is not
			   in the Skyline set
			*/
			if(i != y){
				if(dominance(&points[i * d], &points[y * d], d)){
					is_skyline_point = false;                                            
				}
			}
		}
		/* Copy the results on the device global memory */
		S[y] = is_skyline_point;
	}
}

int main(int argc, char* argv[]){
   	/* Allocate memory to store the number of points, them dimension and the points */
	int* D = (int*) malloc(sizeof(int));
    int* N = (int*) malloc(sizeof(int));
    double* points = build_matrix(stdin, N, D);

	/* - Define the matrix dimension, 
	   - Allocate space on the device global memory 
	   - Copy the array points on the allocated space
	 */
	const size_t size = (*N) * (*D) * sizeof(double);
    double* d_points;
	cudaSafeCall(hipMalloc((void**)&d_points, size));
	cudaSafeCall(hipMemcpy(d_points, points, size, hipMemcpyHostToDevice));

	/* Allocate space where the kernel function will store the result */
	bool *S, *d_S;
	cudaSafeCall(hipMalloc((void**)&d_S, (*N) * sizeof(bool)));	

	/* Define the block and grid dimensions */
	double avg[1024/32] = {0};
	int num_blocks;

	int device;
	int maxActiveBlocks;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);

	for(int k = 0; k < 10; k++){
		for(int i = 1; i <= 32; i++){
			int y_dim = i * 32;
			dim3 block(1, y_dim);
			//printf("%d\n", ((*N) + y_dim - 1)/y_dim);
			dim3 grid(1, ((*N) + y_dim - 1)/y_dim);
			
			/* - Kernel function call to determine the Skyline set
			- Wait it completion to calculate the time spent 
			- Check for errors occurred in the GPU
			 */
		
			double t_kernel_start = hpc_gettime();	
			skyline<<<grid, block>>>(d_points, d_S, *N, *D);
			hipDeviceSynchronize();
			double t_kernel_end = hpc_gettime();

			hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, skyline, y_dim, 0);
			double occupancy = (maxActiveBlocks * y_dim / props.warpSize) / 
				(double)(props.maxThreadsPerMultiProcessor / 
						props.warpSize);
			//printf("y_dim: %d, occ: %f\n", y_dim, occupancy);
			
			avg[i - 1] += t_kernel_end - t_kernel_start;
			//printf("%d %lf\n", y_dim, t_kernel_end - t_kernel_start);
			cudaCheckError();	
		}
	}

	for(int i = 1; i <= 32; i++){
		avg[i - 1] = avg[i - 1] / 10;
		printf("%d %lf\n", i * 32, avg[i - 1]);
	}
		
	/* - Allocate space on the host memory to store the result
	   - Copy the result from device memory to host's
	   - Print the points in the Skyline set 
	*/
	S = (bool*) malloc((*N) * sizeof(bool));
	cudaSafeCall(hipMemcpy(S, d_S, (*N) * sizeof(bool), hipMemcpyDeviceToHost));
	/*for(int i = 0; i < *N; i++){
		if(S[i]){
			for(int k = 0; k < *D; k++){
				printf("%lf ", points[i * (*D) + k]);
			}
			printf("\n");
		}
	}*/

	/* Print the time spent by the kernel to determine the Skyline set */
	//printf("%lf\n", t_kernel_end - t_kernel_start);
	
	/* Free space on device and host heap memory */
	hipFree(d_points);
	free(points);
	free(S);
	free(D);
	free(N);
    return 0;
}
